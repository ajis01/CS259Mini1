#include "hip/hip_runtime.h"
#include <iostream>
#include "dnn.hpp"
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>

using namespace std;

//Define the parameters if not defined externally
#ifndef Nn
  #define Nn 128  // Number of Output Layers
  #define Ni 224  // Number of Input  Layers
#endif

#ifndef Tii
  // Tiling Sizes
  #define Tnn 32  
  #define Tii 32
  //#define Tn 5
  //#define Ti 25
  #define Tn 16
  #define Ti 16
#endif
#define  threadsPerBlock  256
#define  blocksPerGrid    Ni * Nn / threadsPerBlock

//Arrays:
VTYPE synapse[Nn][Ni] __attribute__((aligned(64)));
VTYPE neuron_i[Ni] __attribute__((aligned(64)));
VTYPE neuron_n[Nn] __attribute__((aligned(64))),    neuron_n2[Nn]
__attribute__((aligned(64)));//   neuron_n2_from_dev[Nn] __attribute__((aligned(64)));

void fill_classifier(VTYPE (&synapse)[Nn][Ni], VTYPE (&neuron_i)[Ni], 
    VTYPE (&neuron_n)[Nn],   VTYPE (&neuron_n2)[Nn]) {
  for(int n = 0; n < Nn; ++n) {
    for(int i = 0; i < Ni; ++i) {
      synapse[n][i] = static_cast <float> (rand()) / static_cast <float> (RAND_MAX) - 0.5f;
    }
  }
  for(int i = 0; i < Ni; ++i) {
    neuron_i[i] = static_cast <float> (rand()) / static_cast <float> (RAND_MAX) - 0.5f;
  }
  for(int n = 0; n < Nn; ++n) {
    neuron_n[n] = 0; //i;
    neuron_n2[n] = 0; //i;
  }
}

void classifier_layer(VTYPE (&synapse)[Nn][Ni], VTYPE (&neuron_i)[Ni], VTYPE (&neuron_n)[Nn]) {
  for (int n = 0; n < Nn; n++) {
    VTYPE temp=0;
    for (int i = 0; i < Ni; i++) {
      temp += synapse[n][i] * neuron_i[i];
    }
    neuron_n[n] = transfer(temp);
  }
}

//__global__ void cuda_classifier_layer(VTYPE *dsynapse, VTYPE *dneuron_i, VTYPE *dneuron_n) {
//  for (int n = 0; n < Nn; n++) {
//    VTYPE temp=0;
//    for (int i = 0; i < Ni; i++) {
//      temp += *(dsynapse+ Ni*n + i) * (*(dneuron_i + i));
//    }
//    *(dneuron_n + n) = (temp > 0) ? temp : temp/4;
//  }
//}
__global__ void cuda_classifier_layer(VTYPE *dsynapse, VTYPE *dneuron_i, VTYPE *dneuron_n) {
    __shared__ VTYPE temp[threadsPerBlock];
    int index = blockDim.x*blockIdx.x + threadIdx.x;
      temp[threadIdx.x] = *(dsynapse+ index) * (*(dneuron_i + threadIdx.x));

      __syncthreads();

      if(0 == threadIdx.x)
      {
        VTYPE sum = 0.0;
        for( int i = 0; i < threadsPerBlock; i++ )
          sum += temp[i];
        *(dneuron_n + blockIdx.x) = (sum > 0) ? sum : sum/4;
      }

}

void classifier_layer_blocked(VTYPE (&synapse)[Nn][Ni], VTYPE (&neuron_i)[Ni], 
                              VTYPE (&neuron_n)[Nn]) {
  VTYPE sum[Nn]={0};
  for (int nnn = 0; nnn < Nn; nnn += Tnn) { // tiling for output neurons;
    for (int iii = 0; iii < Ni; iii += Tii) { // tiling for input neurons;
      for (int nn = nnn; nn < nnn + Tnn; nn += Tn) {
        for (int ii = iii; ii < iii + Tii; ii += Ti) {
          // — Original code —
          for (int n = nn; n < nn + Tn; n++) {
            VTYPE sum_sc=0;
            for (int i = ii; i < ii + Ti; i++) {
              sum_sc += (synapse[n][i] * neuron_i[i]);
            }
            sum[n]+=sum_sc;
          }
        }
      }
    }
    for (int nn = nnn; nn < nnn + Tnn; nn++) {
      neuron_n[nn] = transfer(sum[nn]);
    }
  }
}

int main(int argc, char** argv) {
  cout << "initializing arrays\n";

  fill_classifier(synapse,neuron_i,neuron_n,neuron_n2);

  cout << "starting computation\n";

  begin_roi();
  classifier_layer(synapse,neuron_i,neuron_n);
  end_roi();

  cout << "simple version complete!\n";  

  VTYPE * dev_synapse;  
  VTYPE * dev_neuron_i;  
  VTYPE * dev_neuron_n; 
  VTYPE * neuron_n2_from_dev; 
  neuron_n2_from_dev = (VTYPE*) malloc(Nn*sizeof(VTYPE));

  hipError_t err = hipSuccess;
  err = hipMalloc(&dev_synapse,  Ni*Nn*sizeof(VTYPE));
  if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to allocate device synapse (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }

  err = hipMalloc(&dev_neuron_i, Ni*sizeof(VTYPE));
  if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to allocate device neuron_i (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }

  err = hipMalloc(&dev_neuron_n, Nn*sizeof(VTYPE));
  if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to allocate device neuron_n (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }

  err = hipMemcpy(dev_synapse, synapse, Ni*Nn*sizeof(VTYPE), hipMemcpyHostToDevice);

  if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to copy synapse from host to device (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }

  err = hipMemcpy(dev_neuron_i, neuron_i, Ni*sizeof(VTYPE), hipMemcpyHostToDevice);

  if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to copy neuron_i from host to device (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }

  err = hipMemcpy(dev_neuron_n, neuron_n, Nn*sizeof(VTYPE), hipMemcpyHostToDevice);

  if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to copy neuron_n from host to device (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }

  cout << "start cuda simple version complete!\n";  
  begin_roi();
  cuda_classifier_layer <<< blocksPerGrid, threadsPerBlock >>> (dev_synapse,dev_neuron_i,dev_neuron_n);
  err = hipMemcpy(neuron_n2_from_dev, dev_neuron_n, Nn*sizeof(VTYPE), hipMemcpyDeviceToHost);

  if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to copy neuron_n2_from_dev from device to host (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }
  end_roi();

  cout << "cuda simple version complete!\n";  

  begin_roi();
  classifier_layer_blocked(synapse,neuron_i,neuron_n2);  
  end_roi();

  cout << "blocked computation complete!\n";  

  compare(neuron_n,neuron_n2,Nn);
  compare(neuron_n2_from_dev,neuron_n2,Nn);

  cout << "done\n";
  
}

