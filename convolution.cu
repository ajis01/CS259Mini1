#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include "dnn.hpp"
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>

using namespace std;

//Define the parameters if not defined externally
#ifndef Sy
  #define Sy 1
  #define Sx 1
#endif

#ifndef Tnn
  //Tiling Sizes
  #define Tnn 32
  #define Tn  16
  #define Ti  16
  
  #define Ty  8
  #define Tx  8
#endif

#define NYPAD (Ny+Ky)
#define NXPAD (Nx+Kx)

#define NYSCL (Ny/Sy)
#define NXSCL (Nx/Sx)

#define SYNAPSE_SIZE (1L*Ky*Kx*Nn*Ni)

#define  threadsPerBlock  256
#define  blocksPerGrid    (Ni + threadsPerBlock - 1) / threadsPerBlock
#define BATCH 4

VTYPE (*synapse)[Ky][Kx][Nn][Ni];
VTYPE  (*neuron_i)[NYPAD][NXPAD][Ni];
VTYPE  (*neuron_n)[NYSCL][NXSCL][Nn];
VTYPE  (*neuron_n_from_dev)[NYSCL][NXSCL][Nn];
VTYPE  (*neuron_n_from_dev1D)[NYSCL][NXSCL][Nn];
VTYPE  (*neuron_n_from_dev2D)[NYSCL][NXSCL][Nn];
VTYPE (*neuron_n2)[NYSCL][NXSCL][Nn];

VTYPE (*dev_synapse)[Ky][Kx][Nn][Ni];
VTYPE  (*dev_neuron_i)[NYPAD][NXPAD][Ni];
VTYPE  (*dev_neuron_n)[NYSCL][NXSCL][Nn];
VTYPE  (*dev_neuron_n1D)[NYSCL][NXSCL][Nn];
VTYPE  (*dev_neuron_n2D)[NYSCL][NXSCL][Nn];

void fill_convolution_shared_simple(VTYPE (&synapse)[Ky][Kx][Nn][Ni], 
                                    VTYPE (&neuron_i)[NYPAD][NXPAD][Ni]) {
  for(int yy = 0; yy < Ky; ++yy) {
    for(int xx = 0; xx < Kx; ++xx) {
      for(int nn = 0; nn < Nn; ++nn) {
        for(int ni = 0; ni < Ni; ++ni) {
          synapse[yy][xx][nn][ni] = static_cast <float> (rand()) / static_cast <float> (RAND_MAX) - 0.5f;
        } } } }
  for(int yy = 0; yy < NYPAD; ++yy) {
    for(int xx = 0; xx < NXPAD; ++xx) {      
      for(int ni = 0; ni < Ni; ++ni) {
        neuron_i[yy][xx][ni] = static_cast <float> (rand()) / static_cast <float> (RAND_MAX) - 0.5f;
  }  }  }
}

//std::pair<int,int> convolution_layer_blocked(
void convolution_layer_blocked(
                              VTYPE (&synapse)[Ky][Kx][Nn][Ni], 
                              VTYPE (&neuron_i)[NYPAD][NXPAD][Ni], 
                              VTYPE (&neuron_n)[NYSCL][NXSCL][Nn]) {
  VTYPE sum[Nn]={0};

  for (int yy = 0; yy < Ny; yy += Ty) {
    for (int xx = 0; xx < Nx; xx += Tx) {
      for (int nnn = 0; nnn < Nn; nnn += Tnn) {
        int yout = yy/Sy;
        for (int y = yy; y < yy + Ty; y += Sy) { // tiling for y;
          int xout = xx/Sx;

          for (int x = xx; x < xx + Tx; x += Sx) { // tiling for x;

            for (int nn = nnn; nn < nnn + Tnn; nn += Tn) {
              for (int n = nn; n < nn + Tn; n++) {
                sum[n] = 0;
              }

              for (int ky = 0; ky < Ky; ky++) {  // sliding window;
                for (int kx = 0; kx < Kx; kx++) {

                  int ii = 0;
                  VTYPE sum_sc;

                  for (; ii < Ni -Ti+1; ii += Ti) {
                    for (int n = nn; n < nn + Tn; n++) {
                      sum_sc=0;
                      for (int i = ii; i < ii + Ti; i++) {
                        VTYPE sv = synapse[ky][kx][n][i];
                        VTYPE nv = neuron_i[ky + y][kx + x][i];
                        sum_sc+=sv*nv;
                      }
                      sum[n]+=sum_sc;
                    }
                  }
                }
              }

              //transfer
              for (int n = nn; n < nn + Tn; n++) {
                neuron_n[yout][xout][n] = transfer(sum[n]);
              }
            }
            xout++; 
          }
          yout++;
        }
      }
    }
  }
}

void  convolution_layer(VTYPE (&synapse)[Ky][Kx][Nn][Ni], 
                               VTYPE (&neuron_i)[NYPAD][NXPAD][Ni], 
                               VTYPE (&neuron_n)[NYSCL][NXSCL][Nn]) {
  VTYPE sum[Nn]={0};

  // — Original code — (excluding nn, ii loops)
  int yout = 0;
  for (int y = 0; y < Ny; y += Sy) { // tiling for y;
    int xout = 0;
    for (int x = 0; x < Nx; x += Sx) { // tiling for x;
      for (int nn = 0; nn < Nn; nn += Tn) {
        for (int n = nn; n < nn + Tn; n++) {
          sum[n]=0;
        }

        // sliding window;
        for (int ky = 0; ky < Ky; ky++)
          for (int kx = 0; kx < Kx; kx++)
            for (int n = nn; n < nn + Tn; n++)
              for (int i = 0; i < Ni; i++) {
                VTYPE sv = synapse[ky][kx][n][i];
                VTYPE nv = neuron_i[ky + y][kx + x][i];
                sum[n]+=sv*nv;
              }
        for (int n = nn; n < nn + Tn; n++) {
          neuron_n[yout][xout][n] = transfer(sum[n]);
        }
      }
      xout++; 
    }
    yout++;
  }
}


__global__ void  cuda_convolution_layer(VTYPE (&synapse)[Ky][Kx][Nn][Ni], 
                               VTYPE (&neuron_i)[NYPAD][NXPAD][Ni], 
                               VTYPE (&neuron_n)[NYSCL][NXSCL][Nn]) {

  // — Original code — (excluding nn, ii loops)
  int yout = 0;
  for (int y = 0; y < Ny; y += Sy) { // tiling for y;
    int xout = 0;
    for (int x = 0; x < Nx; x += Sx) { // tiling for x;
      for (int n = 0; n < Nn; n++) {

        // sliding window;
        for (int ky = 0; ky < Ky; ky++)
          for (int kx = 0; kx < Kx; kx++)
              for (int i = 0; i < Ni; i++) {
                VTYPE sv = synapse[ky][kx][n][i];
                VTYPE nv = neuron_i[ky + y][kx + x][i];
                neuron_n[yout][xout][n]+=sv*nv;
              }
        neuron_n[yout][xout][n] = (neuron_n[yout][xout][n]>0) ? neuron_n[yout][xout][n] : (neuron_n[yout][xout][n]/4.0);
      }
      xout++; 
    }
    yout++;
  }
}


__global__ void  cuda_convolution_layer1D(VTYPE (&synapse)[Ky][Kx][Nn][Ni], 
                               VTYPE (&neuron_i)[NYPAD][NXPAD][Ni], 
                               VTYPE (&neuron_n)[NYSCL][NXSCL][Nn]) {

  __shared__ VTYPE temp[threadsPerBlock];
  int index = blockDim.x*blockIdx.x + threadIdx.x;
  // — Original code — (excluding nn, ii loops)
  int yout = 0;
  if(index < Ni)
  {
    for (int y = 0; y < Ny; y += Sy) { // tiling for y;
      int xout = 0;
      for (int x = 0; x < Nx; x += Sx) { // tiling for x;
        for (int n = 0; n < Nn; n++) {

          // sliding window;
          for (int ky = 0; ky < Ky; ky++)
            for (int kx = 0; kx < Kx; kx++){
                  VTYPE sv = synapse[ky][kx][n][index];
                  VTYPE nv = neuron_i[ky + y][kx + x][index];
                  //neuron_n[yout][xout][n]+=sv*nv;
                  temp[threadIdx.x] = sv*nv;
                  __syncthreads();
                  if(0 == threadIdx.x)
                  {
                    VTYPE sum = 0.0;
                    for( int i = 0; i < threadsPerBlock; i++ )
                      sum += temp[i];
                    atomicAdd(&(neuron_n[yout][xout][n]),sum);
                  }
                  __syncthreads();
                }
        }
        xout++; 
      }
      yout++;
    }
  }
}

int main(const int argc, const char** argv) {
  cout << "allocating memory\n";

  synapse   = (VTYPE (*)[Ky][Kx][Nn][Ni])  aligned_malloc(64,  SYNAPSE_SIZE*sizeof(VTYPE));
  neuron_i  = (VTYPE (*)[NYPAD][NXPAD][Ni])aligned_malloc(64,NYPAD*NXPAD*Ni*sizeof(VTYPE));
  neuron_n  = (VTYPE (*)[NYSCL][NXSCL][Nn])aligned_malloc(64,NYSCL*NXSCL*Nn*sizeof(VTYPE));
  neuron_n2 = (VTYPE (*)[NYSCL][NXSCL][Nn])aligned_malloc(64,NYSCL*NXSCL*Nn*sizeof(VTYPE));

  neuron_n_from_dev    = (VTYPE (*)[NYSCL][NXSCL][Nn]) malloc(NYSCL*NXSCL*Nn*sizeof(VTYPE));
  neuron_n_from_dev1D  = (VTYPE (*)[NYSCL][NXSCL][Nn]) malloc(NYSCL*NXSCL*Nn*sizeof(VTYPE));
  neuron_n_from_dev2D  = (VTYPE (*)[NYSCL][NXSCL][Nn]) malloc(NYSCL*NXSCL*Nn*sizeof(VTYPE));

  cout << "initializing arrays\n";

  fill_convolution_shared_simple(*synapse,*neuron_i);


  hipError_t err = hipSuccess;
  err = hipMalloc(&dev_synapse,  Ky*Kx*Nn*Ni*sizeof(VTYPE));
  if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to allocate device synapse (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }

  err = hipMalloc(&dev_neuron_i, NYPAD*NXPAD*Ni*sizeof(VTYPE));
  if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to allocate device neuron_i (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }

  err = hipMalloc(&dev_neuron_n, NYSCL*NXSCL*Nn*sizeof(VTYPE));
  if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to allocate device dev_neuron_n (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }

  err = hipMemcpy(dev_synapse, synapse, Kx*Ky*Ni*Nn*sizeof(VTYPE), hipMemcpyHostToDevice);

  if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to copy synapse from host to device (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }

  err = hipMemcpy(dev_neuron_i, neuron_i, NYPAD*NXPAD*Ni*sizeof(VTYPE), hipMemcpyHostToDevice);

  if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to copy neuron_i from host to device (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }

  cout << "starting computation\n";

  //Simple Version
  begin_roi();
  convolution_layer(*synapse,*neuron_i,*neuron_n);
  end_roi();

  cout << "simple version complete!\n";  

  //cout << "starting cuda simple computation\n";

  ////CUDA Simple Version
  //begin_roi();
  //cuda_convolution_layer<<<1,1>>>(*dev_synapse,*dev_neuron_i,*dev_neuron_n);
  //err = hipMemcpy(neuron_n_from_dev, dev_neuron_n, NYSCL*NXSCL*Nn*sizeof(VTYPE), hipMemcpyDeviceToHost);

  //if (err != hipSuccess)
  //{
  //    fprintf(stderr, "Failed to copy neuron_n_from_dev from device to host (error code %s)!\n", hipGetErrorString(err));
  //    exit(EXIT_FAILURE);
  //}
  ////for (int y = 0; y < NYSCL; y++) 
  ////  for (int x = 0; x < NXSCL; x++)
  ////    for (int n = 0; n < Nn; n++)
  ////      *(neuron_n_from_dev)[y][x][n] = transfer(*(neuron_n_from_dev)[y][x][n]);
  //end_roi();

  //cout << "cuda simple version complete!\n";  

  err = hipMalloc(&dev_neuron_n1D, NYSCL*NXSCL*Nn*sizeof(VTYPE));
  if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to allocate device dev_neuron_n1D (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }

  //CUDA 1D Version
  cout << "starting cuda 1D computation\n";
  begin_roi();
  cuda_convolution_layer1D<<<blocksPerGrid,threadsPerBlock>>>(*dev_synapse,*dev_neuron_i,*dev_neuron_n1D);
  err = hipMemcpy(neuron_n_from_dev1D, dev_neuron_n1D, NYSCL*NXSCL*Nn*sizeof(VTYPE), hipMemcpyDeviceToHost);

  if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to copy neuron_n_from_dev1D from device to host (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }
  transfer_array((VTYPE*)*neuron_n_from_dev1D,NYSCL*NXSCL*Nn);
  end_roi();

  cout << "cuda 1D version complete!\n";  

  //Blocked Version
  begin_roi();
  convolution_layer_blocked(*synapse,*neuron_i,*neuron_n2);
  end_roi();


  cout << "blocked computation complete!\n";  

  compare((VTYPE*)*neuron_n,(VTYPE*)*neuron_n2,NYSCL*NXSCL*Nn);
  compare((VTYPE*)*neuron_n,(VTYPE*)*neuron_n_from_dev1D,NYSCL*NXSCL*Nn);

  cout << "done\n";
}


